#include "hip/hip_runtime.h"
#include "bvh.cuh"

__device__
void swap(tri** src_objects, int a, int b) {
    auto t = src_objects[a];
    src_objects[a] = src_objects[b];
    src_objects[b] = t;
}

__device__
int partition(tri** src_objects, int l, int h, bool(*compare)(const tri*, const tri*)) {
    if(l == h)
        return l;

    tri* x = src_objects[h];
    int i = (l - 1);

    for (int j = l; j < h; j++) {
        if (compare(src_objects[j], x)) {
            i++;
            swap(src_objects, i, j);
        }
    }

    swap(src_objects, i + 1, h);
    return (i + 1);
}

__device__
void quicksort_primitives(tri** src_objects, int start, int end, bool(*compare)(const tri*, const tri*)) {
    // Create an auxiliary stack
    auto stack = new int[end - start + 1];

    // initialize top of stack
    int top = -1;

    // push initial values of l and h to stack
    stack[++top] = start;
    stack[++top] = end;

    // Keep popping from stack while is not empty
    while (top >= 0) {
        // Pop h and l
        end = stack[top--];
        start = stack[top--];

        // Set pivot element at its correct position
        // in sorted array
        int p = partition(src_objects, start, end, compare);

        // If there are elements on left side of pivot,
        // then push left side to stack
        if (p - 1 > start) {
            stack[++top] = start;
            stack[++top] = p - 1;
        }

        // If there are elements on right side of pivot,
        // then push right side to stack
        if (p + 1 < end) {
            stack[++top] = p + 1;
            stack[++top] = end;
        }
    }

    delete[] stack;
}

__device__
bool bvh_node::hit(const ray &r, float min, float max, hit_record &rec) const {

    return is_leaf ? primitive->hit(r, min, max, rec) : bbox.hit(r, min, max);
}


__device__
bool bvh::hit(const ray &r, float min, float max, hit_record &rec) const {
    bool res = false;

    if (is_valid()) {
        res = hit(r, min, max, rec, root);
    }
    return res;
}

__device__
bool bvh::hit(const ray &r, float min, float max, hit_record &rec, const bvh_node * const root) {

    bool hit_anything = false;
    float closest_so_far = max;

    // Allocate traversal stack from thread-local memory,
    // and push NULL to indicate that there are no postponed nodes.
    bvh_node* stack[64];
    bvh_node** stack_ptr = stack;
    *stack_ptr++ = nullptr; // push

    // Traverse nodes starting from the root.
    const bvh_node* node = root;

    if (node->is_leaf) {
        //only one element
        if (node->hit(r, min, closest_so_far, rec)) {
            hit_anything = true;
            closest_so_far = rec.t;
        }
    } else do {
            bvh_node* child_l = node->get_left();
            bvh_node* child_r = node->get_right();

            //TODO: verify that rec is not updated if collision isn't closer than current one
            hit_record temp_rec;
            bool hits_l = child_l != nullptr && (child_l->hit(r, min, closest_so_far, temp_rec));
            if (hits_l && child_l->is_leaf) {
                hit_anything = true;
                closest_so_far = temp_rec.t;
                rec = temp_rec;
            }

            bool hits_r = child_r != nullptr && (child_r->hit(r, min, closest_so_far, temp_rec));
            if (hits_r && child_r->is_leaf) {
                hit_anything = true;
                closest_so_far = temp_rec.t;
                rec = temp_rec;
            }

            // Query overlaps an internal node => traverse.
            bool traverse_l = child_l != nullptr && (hits_l && !child_l->is_leaf);
            bool traverse_r = child_r != nullptr && (hits_r && !child_r->is_leaf);

            if (!traverse_l && !traverse_r)
                node = *--stack_ptr; // pop
            else {
                node = traverse_l ? child_l : child_r;
                if (traverse_l && traverse_r)
                    *stack_ptr++ = child_r; //push
            }

        } while (node != nullptr);

    return hit_anything;

}

__device__
void bvh::to_shared(bvh_node* shared_mem, const size_t& shared_mem_size) const {
    //breadth first traverse of bvh to copy higher nodes into shared memory
    shared_mem[0] = *root;
    size_t queue_start = 0;
    size_t queue_end = 1;

    //iterate over node queue, break loop if shared memory is full
    while (queue_end < shared_mem_size && queue_start < queue_end) {
        bvh_node* current = &shared_mem[queue_start];
        
        //check if left node exists
        if (current->left != nullptr) {
            //move node to shared memory
            shared_mem[queue_end] = *(current->left);
            //update pointer to child of current node
            current->left = &shared_mem[queue_end];

            queue_end++;
        }

        //check if right node exists, check for out of bound access in case shared memory is full
        if (queue_end < shared_mem_size && current->right != nullptr) {
            //move node to shared memory
            shared_mem[queue_end] = *(current->right);
            //update pointer to child of current node
            current->right = &shared_mem[queue_end];
            queue_end++;
        }

        //process next node
        queue_start++;
    }
}


__device__ bool bvh::build_bvh(tri** src_objects, size_t list_size, hiprandState* local_rand_state) {

    //dynamic allocation, I do not care about performance during BVH construction anyway since it is only built once
    auto stack = new stack_item[MAX_DEPTH];
    auto node_stack = new  bvh_node*[MAX_DEPTH];

    int tos = -1;
    int n_nodes = 0;
    root = new bvh_node(false);
    tos++;
    n_nodes++;
    stack[tos] = stack_item(0, list_size);
    node_stack[tos] = root;

    while(tos >= 0) {
        //pop
        stack_item current = stack[tos];
        bvh_node* node = node_stack[tos];
        tos--;

        size_t current_span = current.end - current.start;
        if (current_span > 0) {
            bool is_leaf = (current_span == 1);

            if (is_leaf) {
                //create leaf node
                node->is_leaf = true;
                node->left = nullptr;
                node->right = nullptr;
                node->primitive = src_objects[current.start];
            } else {
                int axis = cuda_random_int(0, 2, local_rand_state);
                auto comparator = (axis == 0) ? box_x_compare: ((axis == 1) ? box_y_compare: box_z_compare);

                if (current_span == 2) {
                    //create left and right leaves
                    if (comparator(src_objects[current.start], src_objects[current.start + 1])) {
                        //left
                        node->left = new bvh_node(true);
                        node->left->primitive = src_objects[current.start];

                        //right
                        node->right = new bvh_node(true);
                        node->right->primitive = src_objects[current.start + 1];
                    } else {
                        //left
                        node->left = new bvh_node(true);
                        node->left->primitive = src_objects[current.start+1];

                        //right
                        node->right = new bvh_node(true);
                        node->right->primitive = src_objects[current.start];
                    }

                } else {

                    //sort hittables based on selected axis
                    quicksort_primitives(src_objects, int(current.start), int(current.end - 1), comparator);

                    //more than 2 nodes, create left and right nodes, connect them to current node and push them onto stack
                    node->left = new bvh_node(false);
                    node->right = new bvh_node(false);

                    auto mid = current.start + current_span / 2;

                    //push onto stack

                    //push left child
                    tos++;
                    if (tos >= MAX_DEPTH) {
                        delete[] node_stack;
                        delete[] stack;
                        return false;
                    }

                    stack[tos] = stack_item(current.start, mid);
                    node_stack[tos] = node->left;

                    //push right child
                    tos++;
                    if (tos >= MAX_DEPTH) {
                        delete[] node_stack;
                        delete[] stack;
                        return false;
                    }

                    stack[tos] = stack_item(mid, current.end);
                    node_stack[tos] = node->right;

                    n_nodes+=2;
                }
            }
        }

    }

    size = n_nodes;
    delete[] node_stack;
    delete[] stack;
    return true;
}

__device__ void bvh::build_nodes_bboxes() {
    
    int bbox_created = 0;
    bvh_node* current_node = root;
    
    if(!current_node->is_leaf) { //at least two elements
        int tos = -1;
        auto node_stack = new bvh_node * [size];

        do {
            //traverse to leftmost inner node of subtree
            while (!current_node->is_leaf) {
                node_stack[++tos] = current_node;
                current_node = current_node->left;
            }

            //peek element on tos
            bvh_node* top_node = node_stack[tos];

            if (top_node->right->is_leaf || top_node->right->bbox.isValid()) {
                //compute bbox based on children
                bbox_created++;
                top_node->create_bbox();
                tos--; //actual pop
            }
            else {
                current_node = top_node->right;
            }

        } while (tos >= 0 || !current_node->is_leaf);

        delete[] node_stack;
    }

    
}
