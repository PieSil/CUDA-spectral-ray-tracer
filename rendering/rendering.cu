#include "hip/hip_runtime.h"
//
// Created by pietr on 09/04/2024.
//

#include "rendering.cuh"

__device__
void renderer::ray_bounce(ray &r, const float *background_emittance_spectrum, bvh** bvh, const uint bounce_limit, hiprandState *local_rand_state) {


    hit_record rec;
    //ray cur_ray = r;

    for (int n_bounces = 0; n_bounces < bounce_limit; n_bounces++) {

        if (!(*bvh)->hit(r, 0.0f, FLT_MAX, rec)) {
            r.mul_spectrum(background_emittance_spectrum);

            return; //background * attenuation;
        }

        if (!rec.mat->scatter(r, rec, local_rand_state)) {
            return;
        }
    }

    for(int i = 0; i < N_RAY_WAVELENGTHS; i++) {
        r.power_distr[i] = 0.0f;
    }
}

__device__
point3 renderer::defocus_disk_sample(vec3 camera_center, vec3 defocus_disk_u, vec3 defocus_disk_v, hiprandState* local_rand_state) {
    // Returns a random point in the camera defocus disk.
    auto p = random_in_unit_disk(local_rand_state);
    return camera_center + (p[0] * defocus_disk_u) + (p[1] * defocus_disk_v);
}

__device__
vec3 renderer::pixel_sample_square(const vec3 pixel_delta_u, const vec3 pixel_delta_v, hiprandState* local_rand_state) {
    // Returns a random point in the square surrounding a pixel at the origin.

    auto px = -0.5f + cuda_random_float(local_rand_state);
    auto py = -0.5f + cuda_random_float(local_rand_state);
    return (px * pixel_delta_u) + (py * pixel_delta_v);
};

__device__
vec3 renderer::pixel_stratified_sample_square(const uint sample_x, const uint sample_y, const float recip_sqrt_spp, const vec3 pixel_delta_u, const vec3 pixel_delta_v, hiprandState* local_rand_state) {
    // Returns a random point in the square surrounding a pixel at the origin.
    float px = -0.5f + recip_sqrt_spp * (float(sample_x) + cuda_random_float(local_rand_state));
    float py = -0.5f + recip_sqrt_spp * (float(sample_y) + cuda_random_float(local_rand_state));
    return (px * pixel_delta_u) + (py * pixel_delta_v);
};

__device__
ray renderer::get_ray(uint i, uint j, const point3 pixel00_loc, const vec3 pixel_delta_u, const vec3 pixel_delta_v,
            const point3 camera_center, const vec3 defocus_disk_u, const vec3 defocus_disk_v, const float defocus_angle,
            hiprandState *local_rand_state) {
    /*
     * Get a randomly sampled camera ray for the pixel at location i,j
     * originating from  a random point on the camera defocus disk
     * NOTE: ray direction is not a unit vector in order to have a simpler and slightly faster code
     */

    auto pixel_center = pixel00_loc + ((float)i * pixel_delta_u) + ((float)j * pixel_delta_v);
    auto pixel_sample = pixel_center + pixel_sample_square(pixel_delta_u, pixel_delta_v, local_rand_state);

    auto ray_origin = (defocus_angle <= 0.0f) ? camera_center : defocus_disk_sample(camera_center,
                                                                                    defocus_disk_u,
                                                                                    defocus_disk_v,
                                                                                    local_rand_state);
    auto ray_direction = pixel_sample - ray_origin;

    return ray(ray_origin, ray_direction, local_rand_state);
};

__device__
ray renderer::get_ray_stratified_sample(uint i, uint j,
                              const point3 pixel00_loc,
                              const vec3 pixel_delta_u,
                              const vec3 pixel_delta_v,
                              const uint sample_x,
                              const uint sample_y,
                              const float recip_sqrt_spp,
                              const point3 camera_center,
                              const float defocus_angle,
                              const vec3 defocus_disk_u,
                              const vec3 defocus_disk_v,
                              hiprandState* local_rand_state) {
    /*
     * Get a randomly sampled camera ray for the pixel at location i,j
     * originating from  a random point on the camera defocus disk
     * NOTE: ray direction is not a unit vector in order to have a simpler and slightly faster code
     */

    auto pixel_center = pixel00_loc + ((float)i * pixel_delta_u) + ((float)j * pixel_delta_v);
    auto pixel_sample = pixel_center + pixel_stratified_sample_square(sample_x, sample_y, recip_sqrt_spp, pixel_delta_u, pixel_delta_v, local_rand_state);

    auto ray_origin = (defocus_angle <= 0.0f) ? camera_center : defocus_disk_sample(camera_center,
                                                                                    defocus_disk_u,
                                                                                    defocus_disk_v,
                                                                                    local_rand_state);
    auto ray_direction = pixel_sample - ray_origin;

    return ray(ray_origin, ray_direction, local_rand_state);
};

__global__
void render_init(int max_x, int max_y, hiprandState* rand_state) {
    uint i = threadIdx.x + blockIdx.x * blockDim.x;
    uint j = threadIdx.y + blockIdx.y * blockDim.y;

    
    if((i >= max_x) || (j >= max_y))
        return;
    

    uint thread_index = j * max_x + i;

    //Each thread gets same seed, a different sequence number, no offset
    //hiprand_init(1984, pixel_index, 0, &rand_state[thread_index]);

    //Each thread gets different seed, same sequence number, no offset
    hiprand_init(1984+thread_index, 0, 0, &rand_state[thread_index]);
}

__device__
void save_to_fb(color pixel_color, uint pixel_index, uint samples_per_pixel, vec3* fb) {
    fb[pixel_index] = expand_sRGB(XYZ_to_sRGB(pixel_color / float(samples_per_pixel), reinterpret_cast<const float*>(dev_d65_XYZ_to_sRGB)));
    //fb[pixel_index] = pixel_color/float(samples_per_pixel);
}

__global__
void
spectral_render_kernel(vec3 *fb, bvh **bvh, uint width, uint height, uint offset_x, uint offset_y, camera_data cam_data, float *background_spectrum,
                       const uint samples_per_pixel, const uint bounce_limit, hiprandState *rand_state) {

    uint i = threadIdx.x + blockIdx.x * blockDim.x; //col idx
    uint j = threadIdx.y + blockIdx.y * blockDim.y; //row idx

    extern __shared__ char array[];

    if((i >= width) || j >= height)
        return;

    uint pixel_index = j*width + i;

    //INITIALIZE SHARED MEMORY HERE IF NEEDED
    uint thread_in_block_idx = threadIdx.x*blockDim.y + threadIdx.y;
    float* sh_background_spectrum = (float*)array;
    if (thread_in_block_idx == 0) {
        for(int k = 0; k < N_CIE_SAMPLES; k++) {
            sh_background_spectrum[k] = background_spectrum[k];
        }
    }
//    uint thread_in_block_idx = threadIdx.x*blockDim.y + threadIdx.y;
//    int *light_indices = (int *) array;

//    if (thread_in_block_idx == 0) {
//        int shadow_ray_iterations = int(min(l_list->n_lights, l_list->max_rays));
//
//        for (int offset = 0; offset < blockDim.x * blockDim.y * shadow_ray_iterations; offset += shadow_ray_iterations) {
//            for (int k = 0; k < shadow_ray_iterations; k++) {
//                light_indices[offset + k] = k;
//            }
//        }
//
//        /*
//         * DEBUG PURPOSES
//        if (pixel_index == 0) {
//            for (int offset = 0; offset < blockDim.x * blockDim.y * shadow_ray_iterations; offset++) {
//                printf("light indices [%d] = %d\n", offset, light_indices[offset]);
//            }
//        }
//         */
//    }

    __syncthreads();

    hiprandState local_rand_state = rand_state[pixel_index];
    color pixel_color;

    //TODO: one thread per sample?
    for (int k = 0; k < samples_per_pixel; k++) {
        /*
        * trace the ray from camera center to current pixel sample
        * then sum the sample color obtained from the spectrum to current pixel
        */
        ray r = renderer::get_ray(offset_x + i, offset_y + j, cam_data.pixel00_loc, cam_data.pixel_delta_u, cam_data.pixel_delta_v,
                        cam_data.camera_center, cam_data.defocus_disk_u, cam_data.defocus_disk_v,
                        cam_data.defocus_angle, &local_rand_state);

        
        renderer::ray_bounce(r, sh_background_spectrum, bvh, bounce_limit, &local_rand_state);

        pixel_color += dev_spectrum_to_XYZ(r.wavelengths, r.power_distr, N_RAY_WAVELENGTHS);
    }

    //save updated rand state in random state array for future use
    rand_state[pixel_index] = local_rand_state;


    save_to_fb(pixel_color, pixel_index, samples_per_pixel, fb);
    //save_to_fb(pixel_color, pixel_index, samples_per_pixel, fb);
}

__host__
void renderer::assign_cam_data(camera* cam) {
    cam_data = camera_data(cam->getImageWidth(), cam->getImageHeight(), cam->getPixelDeltaU(), cam->getPixelDeltaV(),
        cam->getPixel00Loc(), cam->getDefocusAngle(), cam->getCenter(),
        cam->getDefocusDiskU(), cam->getDefocusDiskV());
}

__host__
void renderer::call_render_kernel(uint width, uint height, uint offset_x, uint offset_y) {

    if (!device_inited) {
        cerr << "Device parameters were not initialized, render aborted" << endl;
        return;
    }

    // clock_t start, stop;
    //start = clock();

    /*
    camera_data cam_data = camera_data(cam->getImageWidth(), cam->getImageHeight(), cam->getPixelDeltaU(), cam->getPixelDeltaV(),
                                       cam->getPixel00Loc(), cam->getDefocusAngle(), cam->getCenter(),
                                       cam->getDefocusDiskU(), cam->getDefocusDiskV());*/

    spectral_render_kernel<<<blocks, threads, shared_mem_size>>>(dev_fb,
                                                                 dev_bvh,
                                                                 width,
                                                                 height,
                                                                 offset_x,
                                                                 offset_y,
                                                                 cam_data,
                                                                 dev_background_spectrum,
                                                                 samples_per_pixel,
                                                                 bounce_limit,
                                                                 dev_rand_state);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
};

__host__
void renderer::init_device_params(dim3 _threads, dim3 _blocks, uint _max_chunk_width, uint _max_chunk_height) {
    // Allocate Frame Buffer
    //vec3* dev_fb = nullptr;
    threads = _threads;
    blocks = _blocks;
    max_chunk_width = _max_chunk_width;
    max_chunk_height = _max_chunk_height;

    //    PREPARE SHARED MEMORY SIZE HERE IF NEEDED
        //    uint* h_n_lights = new uint;
        //    uint* h_max_rays = new uint;
        //    checkCudaErrors(hipMemcpy(h_n_lights, &(l_list->n_lights), sizeof(uint), hipMemcpyDeviceToHost));
        //    checkCudaErrors(hipMemcpy(h_max_rays, &(l_list->max_rays), sizeof(uint), hipMemcpyDeviceToHost));
        //    int shadow_ray_iterations = int(min(*h_n_lights, *h_max_rays));
        //    free(h_n_lights);
        //    free(h_max_rays);

        //    unsigned shared_mem_size = (shadow_ray_iterations*sizeof(int)*threads.x*threads.y*threads.z);
    shared_mem_size = (N_CIE_SAMPLES);

    uint max_num_pixels = max_chunk_width * max_chunk_height;
    checkCudaErrors(hipMalloc((void**)&dev_fb, max_num_pixels * sizeof(vec3)));
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMalloc((void**)&dev_rand_state, max_num_pixels * sizeof(hiprandState)));
    checkCudaErrors(hipGetLastError());

    float h_background_spectrum[N_CIE_SAMPLES];
    checkCudaErrors(hipMalloc((void**)&dev_background_spectrum, N_CIE_SAMPLES * sizeof(float)));
    checkCudaErrors(hipGetLastError());

    srgb_to_illuminance_spectrum(background, h_background_spectrum);
    checkCudaErrors(hipMemcpy(dev_background_spectrum, h_background_spectrum, N_CIE_SAMPLES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render_init<<<blocks, threads>>>(max_chunk_width, max_chunk_height, dev_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto lc = log_context::getInstance();
    lc->add_entry("chunk width", max_chunk_width);
    lc->add_entry("chunk height", max_chunk_height);

    lc->add_entry("chunk total byte size", max_num_pixels * sizeof(vec3));
    lc->add_entry("shared memory byte size", shared_mem_size);
    lc->add_entry("threads x", threads.x);
    lc->add_entry("threads y", threads.y);
    lc->add_entry("threads z", threads.z);

    lc->add_entry("blocks x", blocks.x);
    lc->add_entry("blocks y", blocks.y);
    lc->add_entry("blocks z", blocks.z);

    

    device_inited = true;
}