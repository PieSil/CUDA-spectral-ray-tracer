#include "hip/hip_runtime.h"
//
// Created by pietr on 09/04/2024.
//

#include "rendering.cuh"

__device__
void renderer::ray_bounce(const uint t_in_block_idx, ray& r, const uint bounce_limit, hiprandState* const local_rand_state) {
	extern __shared__ char array[];

	//hit_record* hit_rec = &shared_hit_records[t_in_block_idx];

	//access shared memeory portion representing hit_records array
	uint block_size = blockDim.x * blockDim.y;
	hit_record* volatile hit_rec = (hit_record*)(&array[BVH_NODE_CACHE_SIZE * sizeof(bvh_node) + t_in_block_idx * sizeof(hit_record)]);
	//hit_record hit_rec;

	for (int n_bounces = 0; n_bounces < bounce_limit; n_bounces++) {

		if (!bvh::hit(r, 0.0f, FLT_MAX, *hit_rec, (bvh_node*)(&array[0]))) {
			//access portion of shared memory representing background
			r.mul_spectrum((float*)(&array[BVH_NODE_CACHE_SIZE * sizeof(bvh_node) + (block_size * sizeof(hit_record))]), N_CIE_SAMPLES);
			return;
		}

		if (!hit_rec->mat->scatter(r, *hit_rec, local_rand_state)) {
			return;
		}

	}

	//for (int i = 0; i < N_RAY_WAVELENGTHS; i++) {
	//	r.power_distr[i] = 0.0f;
	//}
	r.valid_wavelengths = 0;

}

__device__
point3 renderer::defocus_disk_sample(vec3 camera_center, vec3 defocus_disk_u, vec3 defocus_disk_v, hiprandState* local_rand_state) {
	// Returns a random point in the camera defocus disk.
	auto p = random_in_unit_disk(local_rand_state);
	return camera_center + (p[0] * defocus_disk_u) + (p[1] * defocus_disk_v);
}

__device__
vec3 renderer::pixel_sample_square(const vec3 pixel_delta_u, const vec3 pixel_delta_v, hiprandState* local_rand_state) {
	// Returns a random point in the square surrounding a pixel at the origin.

	auto px = -0.5f + cuda_random_float(local_rand_state);
	auto py = -0.5f + cuda_random_float(local_rand_state);
	return (px * pixel_delta_u) + (py * pixel_delta_v);
};

__device__
vec3 renderer::pixel_stratified_sample_square(const uint sample_x, const uint sample_y, const float recip_sqrt_spp, const vec3 pixel_delta_u, const vec3 pixel_delta_v, hiprandState* local_rand_state) {
	// Returns a random point in the square surrounding a pixel at the origin.
	float px = -0.5f + recip_sqrt_spp * (float(sample_x) + cuda_random_float(local_rand_state));
	float py = -0.5f + recip_sqrt_spp * (float(sample_y) + cuda_random_float(local_rand_state));
	return (px * pixel_delta_u) + (py * pixel_delta_v);
};

__device__
ray renderer::get_ray(uint i, uint j, const point3 pixel00_loc, const vec3 pixel_delta_u, const vec3 pixel_delta_v,
	const point3 camera_center, const vec3 defocus_disk_u, const vec3 defocus_disk_v, const float defocus_angle,
	hiprandState* local_rand_state) {
	/*
	 * Get a randomly sampled camera ray for the pixel at location i,j
	 * originating from  a random point on the camera defocus disk
	 * NOTE: ray direction is not a unit vector in order to have a simpler and slightly faster code
	 */

	auto pixel_center = pixel00_loc + ((float)i * pixel_delta_u) + ((float)j * pixel_delta_v);
	auto pixel_sample = pixel_center + pixel_sample_square(pixel_delta_u, pixel_delta_v, local_rand_state);


	auto ray_origin = (defocus_angle <= 0.0f) ? camera_center : defocus_disk_sample(camera_center,
		defocus_disk_u,
		defocus_disk_v,
		local_rand_state);
	auto ray_direction = pixel_sample - ray_origin;

	return ray(ray_origin, ray_direction, local_rand_state);
};

__device__
ray renderer::get_ray_stratified_sample(uint i, uint j,
	const point3 pixel00_loc,
	const vec3 pixel_delta_u,
	const vec3 pixel_delta_v,
	const uint sample_x,
	const uint sample_y,
	const float recip_sqrt_spp,
	const point3 camera_center,
	const float defocus_angle,
	const vec3 defocus_disk_u,
	const vec3 defocus_disk_v,
	hiprandState* local_rand_state) {
	/*
	 * Get a randomly sampled camera ray for the pixel at location i,j
	 * originating from  a random point on the camera defocus disk
	 * NOTE: ray direction is not a unit vector in order to have a simpler and slightly faster code
	 */

	auto pixel_center = pixel00_loc + ((float)i * pixel_delta_u) + ((float)j * pixel_delta_v);
	auto pixel_sample = pixel_center + pixel_stratified_sample_square(sample_x, sample_y, recip_sqrt_spp, pixel_delta_u, pixel_delta_v, local_rand_state);

	auto ray_origin = (defocus_angle <= 0.0f) ? camera_center : defocus_disk_sample(camera_center,
		defocus_disk_u,
		defocus_disk_v,
		local_rand_state);
	auto ray_direction = pixel_sample - ray_origin;

	return ray(ray_origin, ray_direction, local_rand_state);
};

__global__
void init_random_states(int max_x, int max_y, hiprandState* rand_state) {
	//uint i = threadIdx.x + blockIdx.x * blockDim.x;
	//uint j = threadIdx.y + blockIdx.y * blockDim.y;

	uint block_size = blockDim.x * blockDim.y;
	uint block_idx = blockIdx.y * gridDim.x + blockIdx.x;
	uint thread_in_block_idx = threadIdx.y * blockDim.x + threadIdx.x;

	uint coalesced_global_idx = thread_in_block_idx + block_size * block_idx;
	//if ((i >= max_x) || (j >= max_y))
	//	return;


	//uint thread_index = j * max_x + i;

	//Each thread gets different seed, same sequence number, no offset
	hiprand_init(1984 + coalesced_global_idx, 0, 0, &rand_state[coalesced_global_idx]);
}

__device__
void save_to_fb(color& pixel_color, const uint coalesced_global_idx, const uint samples_per_pixel, float* fb_r, float* fb_g, float* fb_b) {

	pixel_color = expand_sRGB(XYZ_to_sRGB(pixel_color / float(samples_per_pixel), reinterpret_cast<const float*>(dev_d65_XYZ_to_sRGB)));

	//write color in coalesced fashion
	fb_r[coalesced_global_idx] = pixel_color[0];
	fb_g[coalesced_global_idx] = pixel_color[1];
	fb_b[coalesced_global_idx] = pixel_color[2];
}

__global__
void
spectral_render_kernel(float* fb_r, float* fb_g, float* fb_b, bvh** bvh, uint width, uint height, uint offset_x, uint offset_y, camera_data cam_data, float* background_spectrum,
	const short_uint samples_per_pixel, const short_uint bounce_limit, hiprandState* rand_state) {

	uint i = threadIdx.x + blockIdx.x * blockDim.x; //col idx
	uint j = threadIdx.y + blockIdx.y * blockDim.y; //row idx

	uint pixel_index = j * width + i;

	uint block_size = blockDim.x * blockDim.y;
	uint block_idx = blockIdx.y * gridDim.x + blockIdx.x;

	uint thread_in_block_idx = threadIdx.y * blockDim.x + threadIdx.x;
	uint coalesced_global_idx = thread_in_block_idx + block_size * block_idx;

	//INITIALIZE SHARED MEMORY HERE IF NEEDED

	//ray* shared_rays = (ray*)(&array[BVH_NODE_CACHE_SIZE * sizeof(bvh_node) + (block_size * sizeof(hit_record))]);
	if (thread_in_block_idx == 0) {
		extern __shared__ char array[];

		bvh_node* bvh_node_cache = (bvh_node*)(&array[0]);
		//access shared memory portion representing background spectrum
		float* sh_background_spectrum = (float*)(&array[BVH_NODE_CACHE_SIZE * sizeof(bvh_node) + (block_size * sizeof(hit_record)) /* + (block_size * sizeof(ray))*/]);

		//write values from global memory
		for (int k = 0; k < N_CIE_SAMPLES; k++) {
			sh_background_spectrum[k] = background_spectrum[k];
		}

		//move higher level nodes to shared memory
		if ((*bvh)->is_valid()) {
			(*bvh)->to_shared(bvh_node_cache, BVH_NODE_CACHE_SIZE);
		}
	}


	__syncthreads();

	if (i >= width || j >= height)
		return;

	//coalesced (as much as possible, since hiprandState size is > 4 bytes) read from rand state array
	hiprandState local_rand_state = rand_state[coalesced_global_idx];


	color pixel_color;

	if ((*bvh)->is_valid()) {
		for (short_uint k = 0; k < samples_per_pixel; k++) {
			/*
			* trace the ray from camera center to current pixel sample
			* then sum the sample color obtained from the spectrum to current pixel
			*/

			ray r = renderer::get_ray(offset_x + i, offset_y + j, cam_data.pixel00_loc, cam_data.pixel_delta_u, cam_data.pixel_delta_v,
				cam_data.camera_center, cam_data.defocus_disk_u, cam_data.defocus_disk_v,
				cam_data.defocus_angle, &local_rand_state);

			renderer::ray_bounce(thread_in_block_idx, r, bounce_limit, &local_rand_state);

			pixel_color += dev_spectrum_to_XYZ(r.wavelengths, r.power_distr, N_RAY_WAVELENGTHS, r.valid_wavelengths);
		}
	}

	//coalesced (as much as possible, since hiprandState size is > 4 bytes) write in rand state array for future use
	rand_state[coalesced_global_idx] = local_rand_state;

	save_to_fb(pixel_color, coalesced_global_idx, samples_per_pixel, fb_r, fb_g, fb_b);
}

__host__
void renderer::assign_cam_data(camera* cam) {
	cam_data = camera_data(cam->getImageWidth(), cam->getImageHeight(), cam->getPixelDeltaU(), cam->getPixelDeltaV(),
		cam->getPixel00Loc(), cam->getDefocusAngle(), cam->getCenter(),
		cam->getDefocusDiskU(), cam->getDefocusDiskV());
}

__host__
void renderer::call_render_kernel(short_uint width, short_uint height, short_uint offset_x, short_uint offset_y) {

	if (!device_inited) {
		cerr << "Device parameters were not initialized, render aborted" << endl;
		return;
	}

	// clock_t start, stop;
	//start = clock();

	/*
	camera_data cam_data = camera_data(cam->getImageWidth(), cam->getImageHeight(), cam->getPixelDeltaU(), cam->getPixelDeltaV(),
									   cam->getPixel00Loc(), cam->getDefocusAngle(), cam->getCenter(),
									   cam->getDefocusDiskU(), cam->getDefocusDiskV());*/

	spectral_render_kernel << <blocks, threads, shared_mem_size >> > (dev_fb_r, dev_fb_g, dev_fb_b,
		dev_bvh,
		width,
		height,
		offset_x,
		offset_y,
		cam_data,
		dev_background_spectrum,
		samples_per_pixel,
		bounce_limit,
		dev_rand_state);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
};

__host__
void renderer::init_device_params(const dim3 _threads, const dim3 _blocks, const uint _max_chunk_width, const uint _max_chunk_height) {
	// Allocate Frame Buffer
	//vec3* dev_fb = nullptr;
	threads = _threads;
	blocks = _blocks;
	max_chunk_width = _max_chunk_width;
	max_chunk_height = _max_chunk_height;

	//    PREPARE SHARED MEMORY SIZE HERE

	uint shared_bg_size = N_CIE_SAMPLES * sizeof(float);
	cout << "shared_bg_size is " << shared_bg_size << endl;
	uint node_cache_size = BVH_NODE_CACHE_SIZE * sizeof(bvh_node);
	cout << "node_cache_size is " << node_cache_size << endl;
	uint shared_hit_rec_size = threads.x * threads.y * sizeof(hit_record);
	cout << "shared_hit_rec_size is " << shared_hit_rec_size << endl;
	//uint shared_rays_size = threads.x * threads.y * sizeof(ray);
	//cout << "shared_rays_size is " << shared_rays_size << endl;

	shared_mem_size = shared_bg_size + node_cache_size + shared_hit_rec_size /*+ shared_rays_size*/;
	uint max_num_pixels = max_chunk_width * max_chunk_height;
	//allocate red buffer
	checkCudaErrors(hipMalloc((void**)&dev_fb_r, /*max_num_pixels*/ threads.x * blocks.x * threads.y * blocks.y * sizeof(float)));
	checkCudaErrors(hipGetLastError());

	//allocate green buffer
	checkCudaErrors(hipMalloc((void**)&dev_fb_g,  /*max_num_pixels*/ threads.x * blocks.x * threads.y * blocks.y * sizeof(float)));
	checkCudaErrors(hipGetLastError());

	//allocate blue buffer
	checkCudaErrors(hipMalloc((void**)&dev_fb_b,  /*max_num_pixels*/ threads.x * blocks.x * threads.y * blocks.y * sizeof(float)));
	checkCudaErrors(hipGetLastError());

	//allocate random state
	checkCudaErrors(hipMalloc((void**)&dev_rand_state, /*max_num_pixels */ threads.x * blocks.x * threads.y * blocks.y * sizeof(hiprandState)));
	checkCudaErrors(hipGetLastError());

	//allocate and initialize background spectrum
	float h_background_spectrum[N_CIE_SAMPLES];
	checkCudaErrors(hipMalloc((void**)&dev_background_spectrum, N_CIE_SAMPLES * sizeof(float)));
	checkCudaErrors(hipGetLastError());

	srgb_to_illuminance_spectrum(background, h_background_spectrum);
	checkCudaErrors(hipMemcpy(dev_background_spectrum, h_background_spectrum, N_CIE_SAMPLES * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	//initialize random state
	init_random_states << <blocks, threads >> > (max_chunk_width, max_chunk_height, dev_rand_state);

	//initialize intersection data buffer
	//alloc_intersection_data << <1, 1 >> > (inters_data_buffer, n_streams, _threads.x * _threads.y, _blocks.x * _blocks.y);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	auto lc = log_context::getInstance();
	lc->add_entry("chunk width", max_chunk_width);
	lc->add_entry("chunk height", max_chunk_height);

	lc->add_entry("chunk total byte size", max_num_pixels * sizeof(vec3));
	lc->add_entry("shared memory byte size", shared_mem_size);
	lc->add_entry("threads x", threads.x);
	lc->add_entry("threads y", threads.y);
	lc->add_entry("threads z", threads.z);

	lc->add_entry("blocks x", blocks.x);
	lc->add_entry("blocks y", blocks.y);
	lc->add_entry("blocks z", blocks.z);

	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(spectral_render_kernel));
	cout << "Max threads per block: " << attr.maxThreadsPerBlock << endl;
	cout << "Registers per thread: " << attr.numRegs << endl;

	device_inited = true;
}

__host__
void renderer::clean_device() {
	if (device_inited) {
		//dealloc_intersection_data << <1, 1 >> > (inters_data_buffer, n_streams);
		//checkCudaErrors(hipFree(inters_data_buffer));
		checkCudaErrors(hipFree(dev_rand_state));
		checkCudaErrors(hipFree(dev_fb_r));
		checkCudaErrors(hipFree(dev_fb_b));
		checkCudaErrors(hipFree(dev_fb_g));
		checkCudaErrors(hipFree(dev_background_spectrum));
		checkCudaErrors(hipGetLastError());
	}
}