#include "hip/hip_runtime.h"
//
// Created by pietr on 13/04/2024.
//

#include "scene.cuh"

using namespace scene;

__global__
void create_bvh_kernel(hittable **d_world, size_t world_size, bvh **d_bvh, bool *success) {

    hiprandState _rand_state = hiprandState();
    hiprandState* rand_state = &_rand_state;
    hiprand_init(1984, 0, 0, rand_state);

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *d_bvh = new bvh(d_world, world_size, rand_state);
        *success = (*d_bvh)->is_valid();
    }
}

__global__
void create_world_kernel(uint world_selector, hittable **d_list, material **d_mat_list, int *world_size, int *n_materials,
                    float* dev_sRGBToSpectrum_Data) {

    /*
     * initialize hittables and materials based on a world selector
     */

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        switch(world_selector){
            case 0:
                device_random_world(d_list, d_mat_list, world_size, n_materials);
                break;
            case 1:
                device_quad_world(d_list, d_mat_list);
                break;

            case 2:
                device_simple_light(d_list, d_mat_list);
                break;
            case 3:
                device_cornell_box(d_list, d_mat_list);
                break;
            default:
                //device_simple_light(d_list, d_mat_list);
                device_random_world(d_list, d_mat_list, world_size, n_materials);
        }

        /*
         * precompute reflectance and emittance spectrum
         */

        
        for(int i = 0; i < *n_materials; i++) {
            d_mat_list[i]->compute_albedo_spectrum(dev_sRGBToSpectrum_Data);
            d_mat_list[i]->compute_emittance_spectrum(dev_sRGBToSpectrum_Data);
        }
        

    }
}

__global__ void
free_world_kernel(hittable **d_list, material **d_mat_list, int world_size, int n_materials, bvh **dev_bvh) {

    if (d_list != nullptr && threadIdx.x == 0 && blockIdx.x == 0) {

        for (int i = 0; i < n_materials; i++) {
            delete *(d_mat_list+i);
        }

        for (int i = 0; i < world_size; i++) {
            delete *(d_list+i);
        }

        delete *dev_bvh;
    }
}

__device__
void scene::device_random_world(hittable **d_list, material **d_mat_list, int *world_size, int *n_materials) {
    hiprandState _rand_state = hiprandState();
    hiprandState* rand_state = &_rand_state;
    hiprand_init(1999, 0, 0, rand_state);
    int void_positions = 0;

    for (int a = -HALF_N_RANDOM_I; a < HALF_N_RANDOM_I; a++) {
        for (int b = -HALF_N_RANDOM_J; b < HALF_N_RANDOM_J; b++) {
            auto choose_mat = cuda_random_float(rand_state);
            point3 center(float(a) + 0.9f*cuda_random_float(rand_state), 0.2f, float(b) + 0.9f*cuda_random_float(rand_state));
            int list_idx = (a+HALF_N_RANDOM_I)*2*HALF_N_RANDOM_J + (b+HALF_N_RANDOM_J)-void_positions;

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {

                if (choose_mat < 0.8f) {
                    // diffuse
                    auto albedo = color::random(rand_state) * color::random(rand_state);
                    *(d_mat_list+list_idx) = new material();
                    **(d_mat_list+list_idx) = material::lambertian(albedo);
                    *(d_list+list_idx) = new sphere(center, 0.2f, *(d_mat_list+list_idx));
                } else if (choose_mat < 0.95f) {
                    // metal
                    //auto albedo = color::random(rand_state);
                    auto albedo = color::random(0.5f, 1.0f, rand_state);
                    auto fuzz = cuda_random_float(0.0f, 0.5f, rand_state);
                    *(d_mat_list+list_idx) = new material();
                    **(d_mat_list+list_idx) = material::metallic(albedo, fuzz);
                    *(d_list+list_idx) = new sphere(center, 0.2f, *(d_mat_list+list_idx));
                } else {
                    // glass
                    *(d_mat_list+list_idx) = new material();
                    **(d_mat_list+list_idx) = material::dielectric(1.5);
                    *(d_list+list_idx) = new sphere(center, 0.2f, *(d_mat_list+list_idx));
                }

                /*if (*(d_list+list_idx) == nullptr)
                    printf("list[%d] is nullptr\n");
                if (*(d_mat_list+list_idx) == nullptr)
                    printf("material[%d] is nullptr\n");*/

            } else {
                void_positions++;
            }
        }
    }

    *(d_mat_list+N_RANDOM_MATERIALS-void_positions) = new material();
    **(d_mat_list+N_RANDOM_MATERIALS-void_positions) = material::lambertian(color(0.5, 0.5, 0.5));
    *(d_list+N_RANDOM_SPHERES-void_positions) = new sphere(point3(0,-1000,0), 1000, *(d_mat_list+N_RANDOM_MATERIALS-void_positions));

    *(d_mat_list+N_RANDOM_MATERIALS-void_positions+1) = new material();
    **(d_mat_list+N_RANDOM_MATERIALS-void_positions+1) = material::dielectric(1.5f);
    *(d_list+N_RANDOM_SPHERES-void_positions+1) = new sphere(point3(0, 1, 0), 1.0, *(d_mat_list+N_RANDOM_MATERIALS-void_positions+1));

    *(d_mat_list+N_RANDOM_MATERIALS-void_positions+2) = new material();
    **(d_mat_list+N_RANDOM_MATERIALS-void_positions+2) = material::lambertian(color(0.4, 0.2, 0.1));
    *(d_list+N_RANDOM_SPHERES-void_positions+2) = new sphere(point3(-4, 1, 0), 1.0, *(d_mat_list+N_RANDOM_MATERIALS-void_positions+2));

    *(d_mat_list+N_RANDOM_MATERIALS-void_positions+3) = new material();
    **(d_mat_list+N_RANDOM_MATERIALS-void_positions+3) = material::metallic(color(0.7, 0.6, 0.5), 0.0);
    *(d_list+N_RANDOM_SPHERES-void_positions+3) = new sphere(point3(4, 1, 0), 1.0, *(d_mat_list+N_RANDOM_MATERIALS-void_positions+3));

    if (world_size != nullptr)
        *world_size = *world_size-void_positions;

    if (n_materials != nullptr)
        *n_materials = *n_materials-void_positions;

    delete rand_state;
}

__device__
void scene::device_quad_world(hittable **d_list, material **d_mat_list) {
    d_mat_list[0] = new material();
    *d_mat_list[0] = material::lambertian(color(1.0, 0.2, 0.2));
    d_mat_list[1] = new material();
    *d_mat_list[1] = material::lambertian(color(0.2, 1.0, 0.2));
    d_mat_list[2] = new material();
    *d_mat_list[2] = material::lambertian(color(0.2, 0.2, 1.0));
    d_mat_list[3] = new material();
    *d_mat_list[3] = material::lambertian(color(1.0, 0.5, 0.0));
    d_mat_list[4] = new material();
    *d_mat_list[4]= material::lambertian(color(0.2, 0.8, 0.8));

    d_list[0] = new quad(point3(-3,-2, 5), vec3(0, 0,-4), vec3(0, 4, 0), d_mat_list[0]);
    d_list[1] = new quad(point3(-2,-2, 0), vec3(4, 0,0), vec3(0, 4, 0), d_mat_list[1]);
    d_list[2] = new quad(point3(3,-2, 1), vec3(0, 0,4), vec3(0, 4, 0), d_mat_list[2]);
    d_list[3] = new quad(point3(-2,3, 1), vec3(4, 0, 0), vec3(0, 0, 4), d_mat_list[3]);
    d_list[4] = new quad(point3(-2,-3, 5), vec3(4, 0, 0), vec3(0, 0, -4), d_mat_list[4]);
}

__device__
void scene::device_simple_light(hittable **d_list, material **d_mat_list) {

    d_mat_list[0] = new material();
    *(d_mat_list[0]) = material::lambertian(color(.0f, 1.0f, .0f));

    d_mat_list[1] = new material();
    //*(d_mat_list[1]) = material::metallic(color(.5f, .5f, .5f), .5f);
    //*(d_mat_list[1]) = material::dielectric(1.5f);
    *(d_mat_list[1]) = material::lambertian(color(.1f, .5f, .7f));

    d_mat_list[2] = new material();
    *(d_mat_list[2]) = material::emissive(color(1.0f, 1.0f, 1.0f), 10.0f);

    d_list[0] = new sphere(point3(0, -1000, 0), 1000, d_mat_list[0]);
    d_list[1] = new sphere(point3(0, 2, 0), 2, d_mat_list[1]);
    d_list[2] = new quad(point3(3, 1, -2), vec3(2, 0, 0), vec3(0, 2, 0), d_mat_list[2]);
}

__device__
void scene::device_cornell_box(hittable **d_list, material **d_mat_list) {
    d_mat_list[0] = new material();
    *d_mat_list[0] = material::lambertian(color(.65, .05, .05));
    d_mat_list[1] = new material();
    *d_mat_list[1] = material::lambertian(color(.73, .73, .73));
    d_mat_list[2] = new material();
    *d_mat_list[2] = material::lambertian(color(.12, .45, .15));
    d_mat_list[3] = new material();
    *d_mat_list[3] = material::emissive(color(15, 15, 15));

//    d_list[0] = new quad(point3(555, 0, 0), vec3(0, 555, 0), vec3 (0, 0, 555), d_mat_list[2]);
//    d_list[1] = new quad(point3(0, 0, 0), vec3(0, 555, 0), vec3 (0, 0, 555), d_mat_list[0]);
//    d_list[2] = new quad(point3(343, 554, 332), vec3(-130, 0, 0), vec3 (0, 0, -105), d_mat_list[3]);
//    d_list[3] = new quad(point3(0, 0, 0), vec3(555, 0, 0), vec3 (0, 0, 555), d_mat_list[1]);
//    d_list[4] = new quad(point3(555, 555, 555), vec3(-555, 0, 0), vec3 (0, 0, -555), d_mat_list[1]);
//    d_list[5] = new quad(point3(0, 0, 555), vec3(555, 0, 0), vec3(0, 555, 0), d_mat_list[1]);

}

__host__
void scene::init_world_parameters(uint world_selector, int *world_size_ptr, int *n_materials_ptr) {

    /*
     * select correct parameters (hard-coded) based on world selector
     */

    switch (world_selector) {
        case 0:
            //random world
            *world_size_ptr = RANDOM_WORLD_SIZE;
            *n_materials_ptr = RANDOM_WORLD_MATERIALS;
            break;

        case 1:
            //quads
            *world_size_ptr = 5;
            *n_materials_ptr = 5;
            break;

        case 2:
            *world_size_ptr = 3;
            *n_materials_ptr = 3;
            break;

        case 3:
            *world_size_ptr = 6;
            *n_materials_ptr = 4;
            break;

        default:
            //random world
            *world_size_ptr = RANDOM_WORLD_SIZE;
            *n_materials_ptr = RANDOM_WORLD_MATERIALS;
            break;
    }
}

__host__
camera_builder scene::random_world_cam_builder() {
    float vfov = 20.0f;
    point3 lookfrom = point3(13,2,3);
    point3 lookat = point3(0,0,0);
    vec3 vup = vec3(0,1,0);
    float defocus_angle = 0.6f;
    float focus_dist = 10.0f;
    color background = color(0.70, 0.80, 1.00);

    return camera_builder().
            setAspectRatio(16.0f/9.0f).
            setImageWidth(400).
            setVfov(vfov).
            setLookfrom(lookfrom).
            setVup(vup).
            setLookat(lookat).
            setDefocusAngle(defocus_angle).
            setFocusDist(focus_dist).
            setBackground(background);
}

__host__
camera_builder scene::quad_world_camera_builder() {
    float vfov = 80.0f;
    point3 lookfrom = point3(0,0,9);
    point3 lookat = point3(0,0,0);
    vec3 vup = vec3(0,1,0);
    float defocus_angle = 0.0f;
    float focus_dist = 10.0f;
    color background = color(0.70, 0.80, 1.00);

    return camera_builder().
            setAspectRatio(1.0f).
            setImageWidth(400).
            setVfov(vfov).
            setLookfrom(lookfrom).
            setVup(vup).
            setLookat(lookat).
            setDefocusAngle(defocus_angle).
            setFocusDist(focus_dist).
            setBackground(background);
}

__host__
camera_builder scene::simple_light_camera_builder() {
    float vfov = 20.0f;
    point3 lookfrom = point3(26,3,6);
    //point3 lookfrom = point3(3,26,6);
    point3 lookat = point3(0,2,0);
    vec3 vup = vec3(0,1,0);
    float defocus_angle = 0.0f;
    float focus_dist = 10.0f;
    //color expanded_b_color = color(10.0f, 180.0f, 186.0f);
    //color expanded_b_color = color(255.0f, 255.0f, 255.0f);
    //color background = expanded_b_color/255;
    //color background = color(0.5f, 0.5f, 0.5f);
    color background = color(0.0f, 0.0f, 0.0f);
    //color background = color(1.00, 1.00, 1.00);

    return camera_builder().
            setAspectRatio(16.0f/9.0f).
            setImageWidth(400).
            setVfov(vfov).
            setLookfrom(lookfrom).
            setVup(vup).
            setLookat(lookat).
            setDefocusAngle(defocus_angle).
            setFocusDist(focus_dist).
            setBackground(background);
}

__host__
camera_builder scene::cornell_box_camera_builder() {
    float vfov = 40.0f;
    point3 lookfrom = point3(278,278,-800);
    point3 lookat = point3(278,278,0);
    vec3 vup = vec3(0,1,0);
    float defocus_angle = 0.0f;
    float focus_dist = 10.0f;
    color background = color(0.0f, 0.0f, 0.0f);
    //color background = color(0.70, 0.80, 1.00);

    return camera_builder().
            setAspectRatio(1.0f).
            setImageWidth(600).
            setVfov(vfov).
            setLookfrom(lookfrom).
            setVup(vup).
            setLookat(lookat).
            setDefocusAngle(defocus_angle).
            setFocusDist(focus_dist).
            setBackground(background);
}

__host__
bool scene::create_bvh(hittable** d_world, size_t world_size, bvh** d_bvh) {
    bool h_success;
    bool* d_success;

    checkCudaErrors(hipMalloc((void**)&d_success, sizeof(bool)));
    create_bvh_kernel<<<1, 1>>>(d_world, world_size, d_bvh, d_success);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(&h_success, d_success, sizeof(bool), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_success));
    checkCudaErrors(hipDeviceSynchronize());

    return h_success;
}

__host__
void scene::create_world(hittable **d_list, material **d_mat_list, int *world_size, int *n_materials,
                  float* dev_sRGBToSpectrum_Data) {
    create_world_kernel<<<1, 1>>>(WORLD_SELECTOR, d_list, d_mat_list, world_size, n_materials, dev_sRGBToSpectrum_Data);
}

__host__
void scene::free_world(hittable **d_list, bvh **dev_bvh, material **d_mat_list, int world_size,
                int n_materials) {
    free_world_kernel<<<1, 1>>>(d_list, d_mat_list, world_size, n_materials, dev_bvh);

}

const result scene_manager::init_world() {
        /*
     * Allocate memory on GPU for hittables and materials, initialize their contents based on a world selector
     * then build a BVH
     */

        int *dev_n_materials_ptr = nullptr;
        int *dev_world_size_ptr = nullptr;

        //select the correct values for size and #materials
        init_world_parameters(WORLD_SELECTOR, h_world_size_ptr, h_n_materials_ptr);

        //copy parameters to device memory so they can be modified from device code
        checkCudaErrors(hipMalloc((void **) &dev_n_materials_ptr, sizeof(int)));
        checkCudaErrors(hipMalloc((void **) &dev_world_size_ptr, sizeof(int)));
        checkCudaErrors(hipMemcpy(dev_n_materials_ptr, h_n_materials_ptr, sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dev_world_size_ptr, h_world_size_ptr, sizeof(int), hipMemcpyHostToDevice));

        //allocate space for world
        checkCudaErrors(hipMalloc((void **) &dev_world, *(h_world_size_ptr) * sizeof(hittable *)));
        if (*(h_world_size_ptr) > 0 && dev_world == nullptr) {
            return {false, "Not enough memory on device for dev_list\n"};
        }

        //allocate space for materials
        checkCudaErrors(hipMalloc((void **) &dev_mat_list, (*h_n_materials_ptr) * sizeof(material *)));
        if ((*h_n_materials_ptr) > 0 && dev_mat_list == nullptr) {
            return {false, "Not enough memory on device for dev_mat_list\n"};
        }

        //allocate space for bvh
        checkCudaErrors(hipMalloc((void **) &dev_bvh, sizeof(bvh *)));
        if (dev_bvh == nullptr) {
            return {false, "Not enough memory on device for BVH\n"};
        }

        //copy constant to device global memory (cannot use constant memory since table is too big)
        float *dev_ColorToSpectrum_Data;
        checkCudaErrors(hipMalloc((void **) &dev_ColorToSpectrum_Data, 3 * 64 * 64 * 64 * 3 * sizeof(float)));
        if (dev_ColorToSpectrum_Data == nullptr) {
            return { false, "Not enough memory on device for dev_ColorToSpectrum_Data\n" };
        }
        checkCudaErrors(
                hipMemcpy(dev_ColorToSpectrum_Data, sRGBToSpectrumTable_Data, 3 * 64 * 64 * 64 * 3 * sizeof(float),
                           hipMemcpyHostToDevice));

        //build hittables and materials
        create_world(dev_world, dev_mat_list, dev_world_size_ptr, dev_n_materials_ptr, dev_ColorToSpectrum_Data);
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipGetLastError());
        //cleanup
        checkCudaErrors(hipFree(dev_ColorToSpectrum_Data));
        checkCudaErrors(hipMemcpy(h_world_size_ptr, dev_world_size_ptr, sizeof(int), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_n_materials_ptr, dev_n_materials_ptr, sizeof(int), hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(dev_world_size_ptr));
        checkCudaErrors(hipFree(dev_n_materials_ptr));

        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        //build bvh
        bool bvh_valid = create_bvh(dev_world, *h_world_size_ptr, dev_bvh);
        if (!bvh_valid && *h_world_size_ptr > 0) {
            return {false, "Error building BVH\n"};
        }

        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        world_inited = true;
        return {true, "World created"};
}

__host__
void scene_manager::init_camera() {
    switch (WORLD_SELECTOR) {
        case 0:
            cam = random_world_cam_builder().getCamera();
            break;
        case 1:
            cam = quad_world_camera_builder().getCamera();
            break;
        case 2:
            cam = simple_light_camera_builder().getCamera();
            break;
        case 3:
            cam = cornell_box_camera_builder().getCamera();
            break;

        default:
            cam = random_world_cam_builder().getCamera();
            break;
    }

    cam_inited = true;
}

__host__
void scene_manager::destroy_world() {
    free_world(dev_world, dev_bvh, dev_mat_list, *h_world_size_ptr, *h_n_materials_ptr);
    checkCudaErrors(hipFree(dev_world));
    checkCudaErrors(hipFree(dev_bvh));
    checkCudaErrors(hipFree(dev_mat_list));

    free(h_world_size_ptr);
    free(h_n_materials_ptr);
}