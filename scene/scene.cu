#include "hip/hip_runtime.h"
//
// Created by pietr on 13/04/2024.
//

#include "scene.cuh"

using namespace scene;

__global__
void create_bvh_kernel(tri** d_world, size_t world_size, bvh** d_bvh, bool* success) {

	hiprandState _rand_state = hiprandState();
	hiprandState* rand_state = &_rand_state;
	hiprand_init(1984, 0, 0, rand_state);

	if (threadIdx.x == 0 && blockIdx.x == 0) {
		*d_bvh = new bvh(d_world, world_size, rand_state);
		*success = (*d_bvh)->is_valid();
	}
}

__global__
void create_world_kernel(uint world_selector, tri** d_list, material** d_mat_list, int* world_size, int* n_materials, float* dev_sRGBToSpectrum_Data) {

	/*
	 * initialize hittables and materials based on a world selector
	 */

	if (threadIdx.x == 0 && blockIdx.x == 0) {
		switch (world_selector) {
			/*
		case RANDOM:
			device_random_world(d_list, d_mat_list, world_size, n_materials);
			break;
		case QUADS:
			device_quad_world(d_list, d_mat_list);
			break;

		case SIMPLE_LIGHT:
			device_simple_light(d_list, d_mat_list);
			break;
			 */

		case PRISM:
			device_prism_test(d_list, d_mat_list);
			break;

			/*
		case SPHERES:
			device_3_spheres(d_list, d_mat_list);
			break;
			 */

		case TRIS:
			device_tri_world(d_list, d_mat_list);
			break;

		case CORNELL:
		default:
			device_cornell_box(d_list, d_mat_list);
			device_cornell_box(d_list, d_mat_list);
			/*
			device_random_world(d_list, d_mat_list, world_size, n_materials);
			 */
		}

		/*
		 * precompute reflectance and emittance spectrum
		 */



		for (int i = 0; i < *n_materials; i++) {
			d_mat_list[i]->compute_spectral_distr(dev_sRGBToSpectrum_Data);
		}

	}
}

__global__ void
free_world_kernel(tri** d_list, material** d_mat_list, int world_size, int n_materials, bvh** dev_bvh) {

	if (d_list != nullptr && threadIdx.x == 0 && blockIdx.x == 0) {

		for (int i = 0; i < n_materials; i++) {
			delete* (d_mat_list + i);
		}

		for (int i = 0; i < world_size; i++) {
			delete* (d_list + i);
		}

		delete* dev_bvh;
	}
}


//__device__
//void scene::device_random_world(hittable** d_list, material** d_mat_list, int* world_size, int* n_materials) {
//	hiprandState _rand_state = hiprandState();
//	hiprandState* rand_state = &_rand_state;
//	hiprand_init(1999, 0, 0, rand_state);
//	int void_positions = 0;
//
//	for (int a = -HALF_N_RANDOM_I; a < HALF_N_RANDOM_I; a++) {
//		for (int b = -HALF_N_RANDOM_J; b < HALF_N_RANDOM_J; b++) {
//			auto choose_mat = cuda_random_float(rand_state);
//			point3 center(float(a) + 0.9f * cuda_random_float(rand_state), 0.2f, float(b) + 0.9f * cuda_random_float(rand_state));
//			int list_idx = (a + HALF_N_RANDOM_I) * 2 * HALF_N_RANDOM_J + (b + HALF_N_RANDOM_J) - void_positions;
//
//			if ((center - point3(4, 0.2, 0)).length() > 0.9) {
//
//				if (choose_mat < 0.8f) {
//					// diffuse
//					auto albedo = color::random(rand_state) * color::random(rand_state);
//					*(d_mat_list + list_idx) = new material();
//					**(d_mat_list + list_idx) = material::lambertian(albedo);
//					*(d_list + list_idx) = new sphere(center, 0.2f, *(d_mat_list + list_idx));
//				}
//				else if (choose_mat < 0.95f) {
//					// metal
//					//auto albedo = color::random(rand_state);
//					auto albedo = color::random(0.5f, 1.0f, rand_state);
//					auto fuzz = cuda_random_float(0.0f, 0.5f, rand_state);
//					*(d_mat_list + list_idx) = new material();
//					**(d_mat_list + list_idx) = material::metallic(albedo, fuzz);
//					*(d_list + list_idx) = new sphere(center, 0.2f, *(d_mat_list + list_idx));
//				}
//				else {
//					// glass
//					*(d_mat_list + list_idx) = new material();
//					**(d_mat_list + list_idx) = material::dielectric_const(1.5f);
//					*(d_list + list_idx) = new sphere(center, 0.2f, *(d_mat_list + list_idx));
//				}
//
//				/*if (*(d_list+list_idx) == nullptr)
//					printf("list[%d] is nullptr\n");
//				if (*(d_mat_list+list_idx) == nullptr)
//					printf("material[%d] is nullptr\n");*/
//
//			}
//			else {
//				void_positions++;
//			}
//		}
//	}
//
//	*(d_mat_list + N_RANDOM_MATERIALS - void_positions) = new material();
//	**(d_mat_list + N_RANDOM_MATERIALS - void_positions) = material::lambertian(color(0.5, 0.5, 0.5));
//	*(d_list + N_RANDOM_SPHERES - void_positions) = new sphere(point3(0, -1000, 0), 1000, *(d_mat_list + N_RANDOM_MATERIALS - void_positions));
//
//	*(d_mat_list + N_RANDOM_MATERIALS - void_positions + 1) = new material();
//	**(d_mat_list + N_RANDOM_MATERIALS - void_positions + 1) = material::dielectric_const(1.5f);
//	*(d_list + N_RANDOM_SPHERES - void_positions + 1) = new sphere(point3(0, 1, 0), 1.0, *(d_mat_list + N_RANDOM_MATERIALS - void_positions + 1));
//
//	*(d_mat_list + N_RANDOM_MATERIALS - void_positions + 2) = new material();
//	**(d_mat_list + N_RANDOM_MATERIALS - void_positions + 2) = material::lambertian(color(0.4, 0.2, 0.1));
//	*(d_list + N_RANDOM_SPHERES - void_positions + 2) = new sphere(point3(-4, 1, 0), 1.0, *(d_mat_list + N_RANDOM_MATERIALS - void_positions + 2));
//
//	*(d_mat_list + N_RANDOM_MATERIALS - void_positions + 3) = new material();
//	**(d_mat_list + N_RANDOM_MATERIALS - void_positions + 3) = material::metallic(color(0.7, 0.6, 0.5), 0.0);
//	*(d_list + N_RANDOM_SPHERES - void_positions + 3) = new sphere(point3(4, 1, 0), 1.0, *(d_mat_list + N_RANDOM_MATERIALS - void_positions + 3));
//
//	if (world_size != nullptr)
//		*world_size = *world_size - void_positions;
//
//	if (n_materials != nullptr)
//		*n_materials = *n_materials - void_positions;
//
//	delete rand_state;
//}

//__device__
//void scene::device_quad_world(hittable** d_list, material** d_mat_list) {
//	d_mat_list[0] = new material();
//	*d_mat_list[0] = material::lambertian(color(1.0, 0.2, 0.2));
//	d_mat_list[1] = new material();
//	*d_mat_list[1] = material::lambertian(color(0.2, 1.0, 0.2));
//	d_mat_list[2] = new material();
//	*d_mat_list[2] = material::lambertian(color(0.2, 0.2, 1.0));
//	d_mat_list[3] = new material();
//	*d_mat_list[3] = material::lambertian(color(1.0, 0.5, 0.0));
//	d_mat_list[4] = new material();
//	*d_mat_list[4] = material::lambertian(color(0.2, 0.8, 0.8));
//
//	d_list[0] = new quad(point3(-3, -2, 5), vec3(0, 0, -4), vec3(0, 4, 0), d_mat_list[0]);
//	d_list[1] = new quad(point3(-2, -2, 0), vec3(4, 0, 0), vec3(0, 4, 0), d_mat_list[1]);
//	d_list[2] = new quad(point3(3, -2, 1), vec3(0, 0, 4), vec3(0, 4, 0), d_mat_list[2]);
//	d_list[3] = new quad(point3(-2, 3, 1), vec3(4, 0, 0), vec3(0, 0, 4), d_mat_list[3]);
//	d_list[4] = new quad(point3(-2, -3, 5), vec3(4, 0, 0), vec3(0, 0, -4), d_mat_list[4]);
//}

//__device__
//void scene::device_simple_light(hittable** d_list, material** d_mat_list) {
//
//	d_mat_list[0] = new material();
//	*(d_mat_list[0]) = material::lambertian(color(.4f, 1.0f, .2f));
//
//	d_mat_list[1] = new material();
//	//*(d_mat_list[1]) = material::metallic(color(.5f, .5f, .5f), .5f);
//	//*(d_mat_list[1]) = material::lambertian(color(.1f, .5f, .7f));
//	*(d_mat_list[1]) = material::dielectric_const(1.5f);
//
//	d_mat_list[3] = new material();
//	*(d_mat_list[3]) = material::dielectric_const(1.0f / 1.5f);
//	//*(d_mat_list[3]) = material::lambertian(color(1.0, 0.0f, 1.0f));
//
//	d_mat_list[2] = new material();
//	*(d_mat_list[2]) = material::emissive(color(1.0f, 1.0f, 1.0f), 5.0f);
//
//	d_list[0] = new sphere(point3(0, -1000, 0), 1000, d_mat_list[0]); //ground
//	d_list[1] = new sphere(point3(0, 2, 0), 2, d_mat_list[1]); //sphere
//	d_list[3] = new sphere(point3(0, 2, 0), 1.9, d_mat_list[3]); //air bubble
//	d_list[2] = new quad(point3(3, 1, -2), vec3(2, 0, 0), vec3(0, 2, 0), d_mat_list[2]); //light
//}

__device__
void scene::device_cornell_box(tri** d_list, material** d_mat_list) {
	d_mat_list[0] = new material();
	*d_mat_list[0] = material::lambertian(color(.65f, .05f, .05f)); //red
	d_mat_list[1] = new material();
	*d_mat_list[1] = material::lambertian(color(.12f, .45f, .15f)); //green
	d_mat_list[2] = new material();
	*d_mat_list[2] = material::dielectric(dev_flint_glass_b, dev_flint_glass_c);
	d_mat_list[3] = new material();
	*d_mat_list[3] = material::lambertian(color(.73f, .73f, .73f)); //white
	d_mat_list[4] = new material();
	*d_mat_list[4] = material::emissive(color(1.f, 1.f, 1.f), 5.f); //light
	d_mat_list[5] = new material();
	*d_mat_list[5] = material::metallic(color(.5f, .5f, .5f), 0.3f); //metal
	d_mat_list[6] = new material();
	*d_mat_list[6] = material::lambertian(color(.12f, .15f, .45f)); //blue

	tri** bottom_faces = &d_list[0];
	tri** top_faces = &d_list[2];
	tri** back_faces = &d_list[4];
	tri** left_faces = &d_list[6];
	tri** right_faces = &d_list[8];
	tri** light_faces = &d_list[10];

	//walls
	tri_quad bottom = tri_quad(point3(0, 0, 0), vec3(0, 0, 555), vec3(555, 0, 0), d_mat_list[3], bottom_faces);
	tri_quad back = tri_quad(point3(0, 0, 555.f), vec3(0, 555, 0), vec3(555, 0, 0), d_mat_list[3], back_faces);
	tri_quad top = tri_quad(point3(555, 555, 555), vec3(-555, 0, 0), vec3(0, 0, -555), d_mat_list[3], top_faces);
	tri_quad left = tri_quad(point3(555, 0, 0), vec3(0, 0, 555), vec3(0, 555, 0), d_mat_list[1], left_faces);
	tri_quad right = tri_quad(point3(0, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), d_mat_list[6], right_faces);

	//light
	point3 center = vec3(555.f / 2.f, 554.f, 555.f / 2.f);
	float width = 100.f;
	float depth = 100.f;
	float height = 150.f;
	float margin = 1.5f;
	point3 Q = point3((center.x() + width / 2.f), center.y(), (center.z() + depth / 2.f));
	tri_quad light = tri_quad(Q, vec3(-width, 0, 0), vec3(0, 0, -depth), d_mat_list[4], light_faces);
	//tri_quad light = tri_quad(point3(343, 554, 332), vec3(-130, 0, 0), vec3(0, 0, -105), d_mat_list[4], light_faces);

	/*
	hittable** box_1_tris = &d_list[12];
	hittable** box_2_tris = &d_list[24];
	hittable** air_tris = &d_list[36];
	hittable** box_3_tris = &d_list[48];
	*/

	//others
	tri_box box1 = tri_box(point3(0.f, 0.f, 0.f), point3(165.f, 330.f, 165.f), d_mat_list[5], &d_list[12]);
	box1.rotate(degrees_to_radians(25.f), transform::AXIS::Y, false);
	box1.translate(vec3(265.f, 0.f, 295.f));


	tri_box box2 = tri_box(point3(0.f, 0.f, 0.f), point3(165.f, 165.f, 165.f), d_mat_list[0], &d_list[24]);
	box2.rotate(degrees_to_radians(-18.f), transform::AXIS::Y, false);
	box2.translate(vec3(130.f, 0.f, 65.f));



	pyramid pyr(point3(165.f, 166.f, 0.f), vec3(-165.f, 0.f, 0.f), vec3(0.f, 0.f, 165.f), vec3(0.f, 165.f, 0.f), d_mat_list[2], &d_list[36]);
	pyr.rotate(degrees_to_radians(-18.f), transform::AXIS::Y, false);
	pyr.translate(vec3(130.f, 0.f, 65.f));

}

__device__
void scene::device_prism_test(tri** d_list, material** d_mat_list) {

	d_mat_list[0] = new material();
	*d_mat_list[0] = material::lambertian(color(.73, .73, .73)); //white
	d_mat_list[1] = new material();
	*d_mat_list[1] = material::emissive(color(1, 1, 1), 5); //light
	d_mat_list[2] = new material();
	//*d_mat_list[2] = material::normal_test(color(0.1f, 0.1f, 0.7f));
	*d_mat_list[2] = material::dielectric(dev_flint_glass_b, dev_flint_glass_c);
	//*d_mat_list[2] = material::dielectric(dev_flint_glass_b, dev_flint_glass_c); //glass
	//*d_mat_list[2] = material::dielectric(dev_BK7_b, dev_BK7_c); //glass

	tri** bottom_faces = &d_list[0];
	tri** top_faces = &d_list[2];
	tri** back_faces = &d_list[4];
	tri** left_faces = &d_list[6];
	tri** right_faces = &d_list[8];
	tri** light_faces = &d_list[10];

	tri_quad bottom = tri_quad(point3(0, 0, 0), vec3(0, 0, 555), vec3(555, 0, 0), d_mat_list[0], bottom_faces);
	tri_quad back = tri_quad(point3(0, 0, 555.f), vec3(0, 555, 0), vec3(555, 0, 0), d_mat_list[0], back_faces);
	tri_quad top = tri_quad(point3(555, 555, 555), vec3(-555, 0, 0), vec3(0, 0, -555), d_mat_list[0], top_faces);
	tri_quad left = tri_quad(point3(555, 0, 0), vec3(0, 0, 555), vec3(0, 555, 0), d_mat_list[0], left_faces);
	tri_quad right = tri_quad(point3(0, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), d_mat_list[0], right_faces);

	//walls
	/*
	d_list[0] = new quad(point3(555, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), d_mat_list[0]);
	d_list[1] = new quad(point3(0, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), d_mat_list[0]);
	d_list[2] = new quad(point3(0, 0, 0), vec3(0, 0, 555), vec3(555, 0, 0), d_mat_list[0]);
	d_list[3] = new quad(point3(555, 555, 555), vec3(0, 0, -555), vec3(-555, 0, 0), d_mat_list[0]);
	d_list[4] = new quad(point3(0, 0, 555), vec3(555, 0, 0), vec3(0, 555, 0), d_mat_list[0]);
	*/

	//light
	point3 center = vec3(555.f / 2.f, 554.f, 555.f / 2.f);
	//float width = 130.f;
	float width = 100.f;
	//float depth = 105.f;
	float depth = 100.f;
	float height = 150.f;
	float margin = 1.5f;

	point3 Q = point3((center.x() + width / 2.f), center.y(), (center.z() + depth / 2.f));
	//d_list[5] = new quad(Q, vec3(-width, 0, 0), vec3(0, 0, -depth), d_mat_list[1]); //light
	tri_quad light = tri_quad(Q, vec3(-width, 0, 0), vec3(0, 0, -depth), d_mat_list[1], light_faces);
	/*
	hittable** light_back_sides = &d_list[12];
	hittable** light_front_sides = &d_list[14];
	hittable** light_right_sides = &d_list[16];
	hittable** light_left_sides = &d_list[18];

	tri_quad light_back(point3(Q.x() + margin, Q.y() + 2.f, (Q.z() - margin) - (depth + 2.f * margin)), vec3(0, -height, 0), vec3(-(width + 2.f * margin), 0, 0), d_mat_list[0], light_back_sides);
	tri_quad light_front(point3(Q.x() + margin, Q.y() + 2.f, Q.z() - margin), vec3(-(width + 2.f * margin), 0, 0), vec3(0, -height, 0), d_mat_list[0], light_front_sides);
	tri_quad light_right(point3((Q.x() + margin), Q.y(), Q.z() - margin), vec3(0.f, -height, 0), vec3(0.f, 0.f, -(depth + 2.f * margin)), d_mat_list[0], light_right_sides);
	tri_quad light_left(point3(Q.x() + margin - (width + 2.f * margin), Q.y(), Q.z() - margin), vec3(0.f, 0.f, -(depth + 2.f * margin)), vec3(0.f, -100.f, 0), d_mat_list[0], light_left_sides);
	*/

	/*
	d_list[6] = new quad(point3(Q.x()+margin, Q.y()+2.f, (Q.z()-margin) - (depth+2.f*margin)), vec3(0, -height, 0), vec3(-(width+2.f*margin), 0, 0), d_mat_list[0]); //back wall
	d_list[7] = new quad(point3(Q.x()+margin, Q.y()+2.f, Q.z()-margin), vec3(-(width+2.f*margin), 0, 0), vec3(0, -height, 0), d_mat_list[0]); //front wall
	d_list[8] = new quad(point3((Q.x()+margin) , Q.y(), Q.z()-margin), vec3(0.f, -height, 0), vec3(0.f, 0.f, -(depth + 2.f*margin)), d_mat_list[0]); //right wall
	d_list[9] = new quad(point3(Q.x()+margin - (width + 2.f*margin), Q.y(), Q.z() - margin), vec3(0.f, 0.f, -(depth + 2.f * margin)), vec3(0.f, -100.f, 0), d_mat_list[0]); //left wall
	*/

	tri** prism_sides = &d_list[12];
	float prism_width = 165.f;
	float prism_height = 200.f;
	//tri_box prism = tri_box(point3(0.f, 0.f, 0.f), point3(165.f, 165.f, 165.f), d_mat_list[2], prism_sides);
	// Q = (center - width/2.f, center.y - 1.f, center.z - prism_height);
	// u = (0.f, -prims_width, 0.f)
	// v = ((prism_width * sqrt(3.f)) / 2.f, prims_width/2.f, 0.f)
	// w = (0.f, 0.f, prism_height);
	// 
	//prism p = prism(point3(0.f, 0.f, 0.f), point3(prism_width, 0.f, 0.f), point3(prism_width / 2.f, 0.f, (prism_width * sqrt(3.f)) / 2.f), point3(0.f, -200.f, 0.f), d_mat_list[2], prism_sides);
	/*NICE*/ prism p = prism(point3(center.x() - width / 2.f, center.y() - 1.f, center.z() - prism_height / 2.f), point3(0.f, -prism_width, 0.f), point3((prism_width * sqrt(3.f)) / 2.f, -prism_width / 2.f, 0.f), point3(0.f, 0.f, 200.f), d_mat_list[2], prism_sides);
	//prism p = prism(point3(center.x() - width / 2.f, center.y() - 1.f, center.z() - prism_height / 2.f), point3(0.f, -prism_width, 0.f), point3((prism_width * sqrt(3.f)) / 2.f, -prism_width / 2.f, 0.f), point3(0.f, 0.f, 200.f), d_mat_list[2], prism_sides);
	p.rotate(degrees_to_radians(10.f), transform::AXIS::Y, true);

	/*
	p.rotate(degrees_to_radians(90.f), transform::AXIS::X, false);
	p.rotate(degrees_to_radians(-30.f), transform::AXIS::Z, false);
	p.rotate(degrees_to_radians(10.f), transform::AXIS::Y, false);
	p.translate(vec3(277.5f - (prism_width/2.f), 0.f , 0.f), false);
	//p.rotate(degrees_to_radians(-45.f), transform::AXIS::Y, false);

	p.translate(vec3(0.f, 450.f, 277.5f), true);
	*/



	/*
	transform::translate_box(prism, vec3(343.f - width - (165.f - width) / 2.f, 553.f - (sqrt(2.0f) * 165.f), 332.f - depth - (165.f - depth) / 2.f), false);
	transform::rotate_box(prism, degrees_to_radians(-45.f), transform::AXIS::Y, false);
	transform::rotate_box(prism, degrees_to_radians(-90.f), transform::AXIS::X, false);
	transform::translate_box(prism, vec3(-50.f, 0.f, 0.f), true);
	*/

}

__device__
void scene::device_tri_world(tri** d_list, material** d_mat_list) {
	d_mat_list[0] = new material();
	*d_mat_list[0] = material::lambertian(color(.65f, .05f, .05f)); //red
	//*d_mat_list[0] = material::normal_test(color(.65f, .05f, .05f)); //red
	d_mat_list[1] = new material();
	//*d_mat_list[1] = material::lambertian(color(.05f, .65f, .05f)); //green
	*d_mat_list[1] = material::lambertian(color(.05f, .65f, .05f)); //green
	d_mat_list[2] = new material();
	//*d_mat_list[2] = material::lambertian(color(.05f, .05f, .65f)); //blue
	//*d_mat_list[2] = material::dielectric_const(1.5f); //glass
	*d_mat_list[2] = material::dielectric(dev_flint_glass_b, dev_flint_glass_c);
	d_mat_list[3] = new material();
	//*d_mat_list[3] = material::lambertian(color(.75f, .75f, .75f)); //white
	*d_mat_list[3] = material::lambertian(color(.75f, .75f, .75f)); //white
	d_mat_list[4] = new material();
	//*d_mat_list[3] = material::lambertian(color(.75f, .75f, .75f)); //white
	*d_mat_list[4] = material::emissive(color(1.f, 1.f, 1.f), 8.f); //light

	tri** bottom_faces = &d_list[0];
	tri** top_faces = &d_list[2];
	tri** back_faces = &d_list[4];
	tri** left_faces = &d_list[6];
	tri** right_faces = &d_list[8];
	tri** light_faces = &d_list[10];

	tri_quad bottom = tri_quad(point3(0, 0, 0), vec3(0, 0, 555), vec3(555, 0, 0), d_mat_list[3], bottom_faces);
	tri_quad back = tri_quad(point3(0, 0, 555.f), vec3(0, 555, 0), vec3(555, 0, 0), d_mat_list[3], back_faces);
	tri_quad top = tri_quad(point3(555, 555, 555), vec3(-555, 0, 0), vec3(0, 0, -555), d_mat_list[3], top_faces);
	tri_quad left = tri_quad(point3(555, 0, 0), vec3(0, 0, 555), vec3(0, 555, 0), d_mat_list[1], left_faces);
	tri_quad right = tri_quad(point3(0, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), d_mat_list[0], right_faces);
	tri_quad light = tri_quad(point3(343, 554, 332), vec3(-130, 0, 0), vec3(0, 0, -105), d_mat_list[4], light_faces);

	/*
	hittable** box_1_tris = &d_list[12];
	hittable** box_2_tris = &d_list[24];
	hittable** air_tris = &d_list[36];
	hittable** box_3_tris = &d_list[48];
	*/

	tri_box box1 = tri_box(point3(0.f, 0.f, 0.f), point3(165.f, 330.f, 165.f), d_mat_list[3], &d_list[12]);
	box1.rotate(degrees_to_radians(15.f), transform::AXIS::Y, false);
	box1.translate(vec3(265.f, 0.f, 295.f));

	tri_box box2 = tri_box(point3(0.f, 0.f, 0.f), point3(165.f, 165.f, 165.f), d_mat_list[2], &d_list[24]);
	box2.rotate(degrees_to_radians(-18.f), transform::AXIS::Y, false);
	box2.translate(vec3(130.f, 0.f, 65.f));

	tri_box air = tri_box(box2, d_mat_list[2], &d_list[36], 1.f);
	air.flip_normals();
}

/*
__device__
void scene::device_3_spheres(tri **d_list, material** d_mat_list) {
	d_mat_list[0] = new material();
	*d_mat_list[0] = material::lambertian(color(.8f, .7f, .0f)); //ground
	d_mat_list[1] = new material();
	*d_mat_list[1] = material::lambertian(color(.1f, .2f, .5f)); //center
	d_mat_list[2] = new material();
	//*d_mat_list[2] = material::lambertian(color(.8f, .8f, .8f)); //left
	*d_mat_list[2] = material::dielectric_const(1.5f);
	d_mat_list[3] = new material();
	*d_mat_list[3] = material::lambertian(color(.8f, .6f, .2f)); //right

	d_mat_list[4] = new material();
	*d_mat_list[4] = material::dielectric_const(1.0f / 1.5f); //air

	d_list[0] = new sphere(point3(0.0, -100.5, -1.0), 100.0, d_mat_list[0]); //ground
	d_list[1] = new sphere(point3(0.0, 0.0, -1.2), 0.5, d_mat_list[1]); //center
	d_list[2] = new sphere(point3(-1.0, 0.0, -1.0), 0.5, d_mat_list[2]); //left
	d_list[3] = new sphere(point3(1.0, 0.0, -1.0), 0.5, d_mat_list[3]); //right
	d_list[4] = new sphere(point3(-1.0, 0.0, -1.0), 0.4, d_mat_list[4]); //air
}
 */

__host__
void scene::init_world_parameters(uint world_selector, int* world_size_ptr, int* n_materials_ptr) {

	/*
	 * select correct parameters (hard-coded) based on world selector
	 */

	switch (world_selector) {
		/*
	case RANDOM:
		//random world
		*world_size_ptr = RANDOM_WORLD_SIZE;
		*n_materials_ptr = RANDOM_WORLD_MATERIALS;
		break;
		 */

		 /*
	 case QUADS:
		 //quads
		 *world_size_ptr = 5;
		 *n_materials_ptr = 5;
		 break;
		  */

		  /*
	  case SIMPLE_LIGHT:
		  //light
		  *world_size_ptr = 4;
		  *n_materials_ptr = 4;
		  break;
		   */

	case PRISM:
		//prism test
		*world_size_ptr = 10 + 2 + 8; //walls + light + prism
		*n_materials_ptr = 3; //white + emission + dielectric
		break;

		/*
	case SPHERES:
		//spheres
		*world_size_ptr = 5;
		*n_materials_ptr = 5;
		break;
		 */

	case TRIS:
		//tris
		*world_size_ptr = 12 + 12 + 12 + 12;
		*n_materials_ptr = 5;
		break;

	case CORNELL:
	default:

		//cornell
		*world_size_ptr = 10 + 2 + 12 + 12 + 6; //walls + light + box1 + box2 + pyramid + prism
		*n_materials_ptr = 7;
		break;
	}
}

//__host__
//camera_builder scene::random_world_cam_builder() {
//	float vfov = 20.0f;
//	point3 lookfrom = point3(13, 2, 3);
//	point3 lookat = point3(0, 0, 0);
//	vec3 vup = vec3(0, 1, 0);
//	float defocus_angle = 0.6f;
//	float focus_dist = 10.0f;
//	color background = color(0.70, 0.80, 1.00);
//
//	return camera_builder().
//		setVfov(vfov).
//		setLookfrom(lookfrom).
//		setVup(vup).
//		setLookat(lookat).
//		setDefocusAngle(defocus_angle).
//		setFocusDist(focus_dist).
//		setBackground(background);
//}

//__host__
//camera_builder scene::quad_world_camera_builder() {
//	float vfov = 80.0f;
//	point3 lookfrom = point3(0, 0, 9);
//	point3 lookat = point3(0, 0, 0);
//	vec3 vup = vec3(0, 1, 0);
//	float defocus_angle = 0.0f;
//	float focus_dist = 10.0f;
//	color background = color(0.70, 0.80, 1.00);
//
//	return camera_builder().
//		setVfov(vfov).
//		setLookfrom(lookfrom).
//		setVup(vup).
//		setLookat(lookat).
//		setDefocusAngle(defocus_angle).
//		setFocusDist(focus_dist).
//		setBackground(background);
//}

//__host__
//camera_builder scene::simple_light_camera_builder() {
//	float vfov = 20.0f;
//	point3 lookfrom = point3(26, 3, 6);
//	//point3 lookfrom = point3(3,26,6);
//	point3 lookat = point3(0, 2, 0);
//	vec3 vup = vec3(0, 1, 0);
//	float defocus_angle = 0.0f;
//	float focus_dist = 10.0f;
//	//color expanded_b_color = color(10.0f, 180.0f, 186.0f);
//	//color expanded_b_color = color(255.0f, 255.0f, 255.0f);
//	//color background = expanded_b_color/255;
//	//color background = color(0.5f, 0.5f, 0.5f);
//	//color background = color(0.7f, 0.7f, 1.0f);
//	color background = color(0.0f, 0.0f, 0.0f);
//	//color background = color(1.00, 1.00, 1.00);
//
//	return camera_builder().
//		setVfov(vfov).
//		setLookfrom(lookfrom).
//		setVup(vup).
//		setLookat(lookat).
//		setDefocusAngle(defocus_angle).
//		setFocusDist(focus_dist).
//		setBackground(background);
//}

__host__
camera_builder scene::cornell_box_camera_builder() {
	float vfov = 40.0f;
	point3 lookfrom = point3(278, 278, -800);
	point3 lookat = point3(278, 278, 0);
	vec3 vup = vec3(0, 1, 0);
	float defocus_angle = 0.0f;
	float focus_dist = 10.0f;
	color background = color(0.0f, 0.0f, 0.0f);
	//color background = color(0.70, 0.80, 1.00);

	return camera_builder().
		setVfov(vfov).
		setLookfrom(lookfrom).
		setVup(vup).
		setLookat(lookat).
		setDefocusAngle(defocus_angle).
		setFocusDist(focus_dist).
		setBackground(background);
}

__host__
camera_builder scene::tris_camera_builder() {
	float vfov = 40.0f;
	point3 lookfrom = point3(278, 278, -800);
	point3 lookat = point3(278, 278, 0);
	vec3 vup = vec3(0, 1, 0);
	float defocus_angle = 0.0f;
	float focus_dist = 10.0f;
	color background = color(0.0f, .0f, 0.f);
	//color background = color(0.7f, .7f, 1.f);

	return camera_builder().
		setVfov(vfov).
		setLookfrom(lookfrom).
		setVup(vup).
		setLookat(lookat).
		setDefocusAngle(defocus_angle).
		setFocusDist(focus_dist).
		setBackground(background);
}

__host__
camera_builder scene::prism_test_camera_builder() {
	float vfov = 40.0f;
	point3 lookfrom = point3(278, 278, -800);
	point3 lookat = point3(278, 278, 0);
	vec3 vup = vec3(0, 1, 0);
	float defocus_angle = 0.0f;
	float focus_dist = 10.0f;
	color background = color(0.0f, 0.0f, 0.0f);
	//color background = color(0.70, 0.80, 1.00);

	return camera_builder().
		setVfov(vfov).
		setLookfrom(lookfrom).
		setVup(vup).
		setLookat(lookat).
		setDefocusAngle(defocus_angle).
		setFocusDist(focus_dist).
		setBackground(background);
}

//__host__
//camera_builder scene::spheres_camera_builder() {
//	float vfov = 80.0f;
//	point3 lookfrom = point3(0, 0, 0);
//	point3 lookat = point3(0, 0, -1);
//	vec3 vup = vec3(0, 1, 0);
//	float defocus_angle = 0.0f;
//	float focus_dist = 10.0f;
//	color background = color(0.7, 0.8, 1.0);
//
//	return camera_builder().
//		setVfov(vfov).
//		setLookfrom(lookfrom).
//		setVup(vup).
//		setLookat(lookat).
//		setDefocusAngle(defocus_angle).
//		setFocusDist(focus_dist).
//		setBackground(background);
//}

__host__
bool scene::create_bvh(tri** d_world, size_t world_size, bvh** d_bvh) {
	bool h_success;
	bool* d_success;

	checkCudaErrors(hipMalloc((void**)&d_success, sizeof(bool)));
	create_bvh_kernel << <1, 1 >> > (d_world, world_size, d_bvh, d_success);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(&h_success, d_success, sizeof(bool), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_success));
	checkCudaErrors(hipDeviceSynchronize());

	return h_success;
}

__host__
void scene::create_world(uint world_selector, tri** d_list, material** d_mat_list, int* world_size, int* n_materials, float* dev_sRGBToSpectrum_Data) {
	create_world_kernel << <1, 1 >> > (world_selector, d_list, d_mat_list, world_size, n_materials, dev_sRGBToSpectrum_Data);
}

__host__
void scene::free_world(tri** d_list, bvh** dev_bvh, material** d_mat_list, int world_size,
	int n_materials) {
	free_world_kernel << <1, 1 >> > (d_list, d_mat_list, world_size, n_materials, dev_bvh);

}

const result scene_manager::init_world() {
	/*
 * Allocate memory on GPU for hittables and materials, initialize their contents based on a world selector
 * then build a BVH
 */

	int* dev_n_materials_ptr = nullptr;
	int* dev_world_size_ptr = nullptr;

	//select the correct values for size and #materials
	init_world_parameters(selected_world, h_world_size_ptr, h_n_materials_ptr);

	//copy parameters to device memory so they can be modified from device code
	checkCudaErrors(hipMalloc((void**)&dev_n_materials_ptr, sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&dev_world_size_ptr, sizeof(int)));
	checkCudaErrors(hipMemcpy(dev_n_materials_ptr, h_n_materials_ptr, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_world_size_ptr, h_world_size_ptr, sizeof(int), hipMemcpyHostToDevice));

	//allocate space for world
	checkCudaErrors(hipMalloc((void**)&dev_world, *(h_world_size_ptr) * sizeof(tri*)));
	if (*(h_world_size_ptr) > 0 && dev_world == nullptr) {
		return { false, "Not enough memory on device for dev_list\n" };
	}

	//allocate space for materials
	checkCudaErrors(hipMalloc((void**)&dev_mat_list, (*h_n_materials_ptr) * sizeof(material*)));
	if ((*h_n_materials_ptr) > 0 && dev_mat_list == nullptr) {
		return { false, "Not enough memory on device for dev_mat_list\n" };
	}

	//allocate space for bvh
	checkCudaErrors(hipMalloc((void**)&dev_bvh, sizeof(bvh*)));
	if (dev_bvh == nullptr) {
		return { false, "Not enough memory on device for BVH\n" };
	}

	//copy constant to device global memory (cannot use constant memory since table is too big)


	float* dev_ColorToSpectrum_Data;
	checkCudaErrors(hipMalloc((void**)&dev_ColorToSpectrum_Data, 3 * 64 * 64 * 64 * 3 * sizeof(float)));
	if (dev_ColorToSpectrum_Data == nullptr) {
		return { false, "Not enough memory on device for dev_ColorToSpectrum_Data\n" };
	}
	checkCudaErrors(
		hipMemcpy(dev_ColorToSpectrum_Data, sRGBToSpectrumTable_Data, 3 * 64 * 64 * 64 * 3 * sizeof(float),
			hipMemcpyHostToDevice));



	//build hittables and materials
	create_world(selected_world, dev_world, dev_mat_list, dev_world_size_ptr, dev_n_materials_ptr, dev_ColorToSpectrum_Data);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());
	//cleanup
	checkCudaErrors(hipFree(dev_ColorToSpectrum_Data));
	checkCudaErrors(hipMemcpy(h_world_size_ptr, dev_world_size_ptr, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_n_materials_ptr, dev_n_materials_ptr, sizeof(int), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(dev_world_size_ptr));
	checkCudaErrors(hipFree(dev_n_materials_ptr));

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	//build bvh
	bool bvh_valid = create_bvh(dev_world, *h_world_size_ptr, dev_bvh);
	if (!bvh_valid && *h_world_size_ptr > 0) {
		return { false, "Error building BVH\n" };
	}

	auto lc = log_context::getInstance();
	lc->add_entry("scene type", sceneIdToStr[selected_world]);
	lc->add_entry("# primitives", *h_world_size_ptr);
	lc->add_entry("# materials", *h_n_materials_ptr);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	world_inited = true;
	return { true, "World created" };
}

__host__
void scene_manager::init_camera() {
	camera_builder cam_builder;
	switch (selected_world) {
		/*
	case RANDOM:
		cam_builder = random_world_cam_builder();
		break;
		 */
		 /*
	 case QUADS:
		 cam_builder = quad_world_camera_builder();
		 break;
		  */
		  /*
	  case SIMPLE_LIGHT:
		  cam_builder = simple_light_camera_builder();
		  break;
		   */

	case PRISM:
		cam_builder = prism_test_camera_builder();
		break;
		/*
	case SPHERES:
		cam_builder = spheres_camera_builder();
		break;
		 */
	case TRIS:
		cam_builder = tris_camera_builder();
		break;

	case CORNELL:
	default:
		cam_builder = cornell_box_camera_builder();
		break;
	}
	auto pm = param_manager::getInstance();
	cam = cam_builder.getCamera();

	auto lc = log_context::getInstance();
	lc->add_entry("image width", cam.getImageWidth());
	lc->add_entry("image height", cam.getImageHeight());

	cam_inited = true;
}

__host__
void scene_manager::destroy_world() {
	free_world(dev_world, dev_bvh, dev_mat_list, *h_world_size_ptr, *h_n_materials_ptr);
	checkCudaErrors(hipFree(dev_world));
	checkCudaErrors(hipFree(dev_bvh));
	checkCudaErrors(hipFree(dev_mat_list));

	free(h_world_size_ptr);
	free(h_n_materials_ptr);
}